#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "ovParticleSystemDX.cuh"

__global__ void
checkParticles(float* lifeTime, bool* alive) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  alive[index] = lifeTime[index] > 0.0f;
}

namespace ovEngineSDK {
  
  void
  ParticleSystemDX::init() {
    
  }

  void
  ParticleSystemDX::update(float delta) {
  }

  void
  ParticleSystemDX::destroy() {
    
  }
}